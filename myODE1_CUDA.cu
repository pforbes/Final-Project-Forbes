#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include "cuda_complex.hpp"
#include <math.h>
#include <random>
#include <chrono>
#include "stopwatch.hpp"
#include <hip/hip_runtime.h>
	
const double b_sigma = 0.01; // normal distribution std deviation for b
/* set b0 values to increment: # points = (MAX-MIN)/INCR+1 */
const double MIN_b0 = 1.515;
const double MAX_b0 = 1.52;
const double INCR_b0 =0.00001;
const int SIZE_b0 = int((MAX_b0-MIN_b0)/INCR_b0)+1;

/* set C values to increment: # points = (MAX-MIN)/INCR+1 */
const double MIN_C = 0.019;
const double MAX_C = 0.020;
const double INCR_C= 0.0001;
const int SIZE_C = int((MAX_C-MIN_C)/INCR_C)+1;
  
/* Set resolution in length along device ('time' for ODE solvers) */
const double L_max = 150.0;
//const double L_inc = 0.001;

const int B_LENGTH = int(L_max) + 1;  // length of the b vector
//const double d = 0;
//const double QC = 1.85*0.25;


__global__ void ODE_Kernel(double* db0_in, double* dC_in,double *db_rand, double* dLength,double* dGain, int num_loops, int b_length)
{
  double dt = 0.001;
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  
  //
  // should implement shared memory method for interpolating b(x)
  //
  
  if (idx < num_loops)
    {
      double b0 = db0_in[idx];
      double C  = dC_in[idx];
      double QC = 0.25*1.85;
      double d = 0.0;
      
      //state initialization
      complex<double> x0(0.0,0.0);
      complex<double> x1(0.0,0.0);
      complex<double> x2(1.0,0.0);

      complex<double> i(0,1);
      
      // Solve ODE using
	for(int n = 0; n < 150000; n++) {

	//[ this should be replace with shared memory above
	// calculate current b 
	int T_low = int(floor(n*dt));
	int T_high = T_low + 1;
	if (T_low >= b_length-1) {
	  T_high = T_low;
	}
	double b =  b0 + db_rand[T_low] + double(n*dt-T_low)*(db_rand[T_high]-db_rand[T_low]);
	
	//]

	
	//[ it seems k and l can be simplified to reduce # of registers used i.e. k1 = 1.5*k0, k2 = 2.25*k0
	complex<double> k0 = dt*x1;
	complex<double> l0 = dt*x2;
	complex<double> m0 = dt*(-i*C*C*C*(4.0*QC*(b+i*d)-1.0)*x0 - 4.0*QC*C*C*x1 - i*C*(b + i*d)*x2);
	complex<double> k1 = dt*(x1+0.5*l0);
	complex<double> l1 = dt*(x2+0.5*m0);
	complex<double> m1 = dt*(-i*C*C*C*(4.0*QC*(b+i*d)-1.0)*(x0+0.5*k0) - 4.0*QC*C*C*(x1+0.5*l0) - i*C*(b + i*d)*(x2+0.5*m0));;
	complex<double> k2 = dt*(x1+0.5*l1);
	complex<double> l2 = dt*(x2+0.5*m1);
	complex<double> m2 = dt*(-i*C*C*C*(4.0*QC*(b+i*d)-1.0)*(x0+0.5*k1) - 4.0*QC*C*C*(x1+0.5*l1) - i*C*(b + i*d)*(x2+0.5*m1));;
	complex<double> k3 = dt*(x1+0.5*l2);
	complex<double> l3 = dt*(x2+0.5*m2);
	complex<double> m3 = dt*(-i*C*C*C*(4.0*QC*(b+i*d)-1.0)*(x0+0.5*k2) - 4.0*QC*C*C*(x1+0.5*l2) - i*C*(b + i*d)*(x2+0.5*m2));;
	x0 = x0 + 1.0/6.0*(k0 + k1+k1 + k2+k2 + k3);
	x1 = x1 + 1.0/6.0*(l0 + l1+l1 + l2+l2 + l3);
	x2 = x2 + 1.0/6.0*(m0 + m1+m1 + m2+m2 + m3);
	//]
	
	// calculate gain(x)
	double GAIN = std::pow(abs( 1.0/(x2 + 4.0*QC*C*C*x0)),2);
	
	// test for best gain
	if ( dGain[idx] < GAIN)
	  {
	    dLength[idx] = n*dt; // store current length if best gain
	    dGain[idx] = GAIN; // store Gain if best gain
	  }
	}
	// change gain to dB units
      	dGain[idx] =  10*std::log10(dGain[idx]);
    }
}


int main(int  argc , char*  argv[]  )
{
  stopwatch<std::milli, float> sw_incl;
  sw_incl.start();
  
  int num_loops = SIZE_b0*SIZE_C;
  // vectors for input data
  double *b0_in = (double*)malloc(sizeof(double)*SIZE_b0);
  double *C_in = (double*)malloc(sizeof(double)*SIZE_C);

  // create b0_in and C_in vectors
  b0_in[0] = MIN_b0;
  C_in[0] = MIN_C;
  for (int b = 1; b < SIZE_b0; b++) {
    b0_in[b] = b0_in[b-1] + INCR_b0;
  }
  for (int c = 1; c < SIZE_C; c++) {
    C_in[c] = C_in[c-1] + INCR_C;
  }

  // vectors for output data
  double *C_out = (double*)malloc(sizeof(double)*num_loops);
  double *b0_out = (double*)malloc(sizeof(double)*num_loops);
  double *b_rand = (double*)malloc(sizeof(double)*B_LENGTH);
  double *gain_dB_out = (double*)malloc(sizeof(double)*num_loops);
  double *length_out = (double*)malloc(sizeof(double)*num_loops);

  // create C_out and b0_out vectors
  int idx = 0;
  for (int b = 0; b<SIZE_b0; b++) {
    for (int c = 0; c<SIZE_C; c++) {
      b0_out[idx] = b0_in[b];
      C_out[idx] = C_in[c];
      idx++;
    }
  }

  // set up random b(x) distribution
  unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
  std::default_random_engine generator (seed);
  std::normal_distribution<double> distribution(0,b_sigma);
  for (int n = 0; n < B_LENGTH; n++) {
    b_rand[n] = distribution(generator);
  }

  std::cout << SIZE_b0 << ' '  << ' ' << SIZE_C << '\n';
  std::cout << b0_in[0] << ' ' << b0_in[SIZE_b0-1] << '\n';
  
  // Allocate device memory 
  double *db0_in, *dC_in, *db_rand, *dLength, *dGain;

  // START TIMER
  hipEvent_t startTime, stopTime;
  float time;
  hipEventCreate(&startTime);
  hipEventCreate(&stopTime);
  hipEventRecord(startTime,0);

  hipMalloc((void**)&db0_in, sizeof(double)*num_loops);
  hipMalloc((void**)&dC_in, sizeof(double)*num_loops);
  hipMalloc((void**)&db_rand, sizeof(double)*B_LENGTH);
  hipMalloc((void**)&dLength, sizeof(double)*num_loops);
  hipMalloc((void**)&dGain, sizeof(double)*num_loops);

  // send all data to GPU 
  hipMemcpy(db0_in, b0_out, sizeof(double)*num_loops, hipMemcpyHostToDevice);
  hipMemcpy(dC_in, C_out, sizeof(double)*num_loops, hipMemcpyHostToDevice);
  hipMemcpy(db_rand, b_rand, sizeof(double)*B_LENGTH, hipMemcpyHostToDevice);
  hipMemset(dGain, 0, sizeof(double)*num_loops);
  hipMemset(dLength, 0, sizeof(double)*num_loops);

  // Call Cuda Kernels to solve ODE
  int num_threads = 256;
  int num_blocks = 22;

  ODE_Kernel<<<num_blocks,num_threads>>>(db0_in,dC_in,db_rand,dLength,dGain, num_loops, B_LENGTH);

  hipMemcpy(length_out, dLength, sizeof(double)*num_loops, hipMemcpyDeviceToHost);
  hipMemcpy(gain_dB_out, dGain, sizeof(double)*num_loops, hipMemcpyDeviceToHost);  
  
  // stop inclusive timer
  hipEventRecord(stopTime,0);
  hipEventSynchronize(stopTime);
  hipEventElapsedTime(&time, startTime, stopTime);
  hipEventDestroy(startTime);
  hipEventDestroy(stopTime);

  // print output data to terminal
  std::cout  << 'C' << '\t' << "b0" << '\t' << "maxLength" << '\t' << "maxGain" << '\t' << "maxGain[dB]" << std::endl;  
  std::cout << "---------------------------------------------------" << std::endl;
  for(int n = 0; n < num_loops; n++) {
    std::cout << C_out[n] << '\t' << b0_out[n] << '\t' << length_out[n] << '\t' << gain_dB_out[n] << '\t'<< n << '\n';
  }   
  std::cout << "time[ms]" << '\t' << "num_loops" << '\t'<< "ms/loop" << '\n';
  std::cout << time << '\t' << num_loops << '\t'<< time/float(num_loops) << '\n';

  // free device memory
  hipFree(db0_in);
  hipFree(dC_in);
  hipFree(db_rand);
  hipFree(dLength);
  hipFree(dGain);

  // free host memory
  free(length_out);
  free(gain_dB_out);
  free(C_in);
  free(C_out);
  free(b0_in);
  free(b0_out);
  free(b_rand);
}
